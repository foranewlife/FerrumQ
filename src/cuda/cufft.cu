
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


void compute_fft(hipfftComplex* input, hipfftComplex* output, int size) {
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, input, output, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}
